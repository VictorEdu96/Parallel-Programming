#include "hip/hip_runtime.h"
/*
 * PARA CORRERLO
 *   export LD_LIBRARY_PATH=/usr/local/cuda/lib
 *   export PATH=$PATH:/usr/local/cuda/bin
 *   nvcc -o prueba prueba.cu -O2 -lc -lm
 *   ./prueba n
*/

/*
 * UNSIGNED INT --> Tipo de dato para enteros, números sin punto decimal. 
 *                  Los enteros sin signo pueden ser tan grandes como 65535 
 *                  y tan pequeños como 0. 
 *                  Son almacenados como 16 bits de información.
 *
 * SIZE_T --> is an unsigned integer type guaranteed to support the longest 
 *            object for the platform you use. It is also the result of the 
 *            sizeof operator.sizeof returns the size of the type in bytes.
 *            So in your context of question in both cases you pass a 
 *            size_t to malloc.
 */

 #include <stdio.h>
 #include <stdlib.h>
 #include <assert.h>
 
 #define NUMBER_THREADS 16
 float elapsed_time_ms;  

 char *dev_mat_in, *dev_mat_out;

//---------------------------------------------------------------------------

 void printMatrix(unsigned int rows, unsigned int cols, char *matrix){
    unsigned int size = rows * cols;
    for (int i = 0; i < size; i++) {
       printf("%i, ", matrix[i]);
       printf(rows % size);
       if(rows % size == 0) {
           printf("\n");
       }
    }
    printf("\n");
}


 /* Kernel code */
 __global__ void transpose_gpu(const char *mat_in, char *mat_out, unsigned int rows, unsigned int cols){
     unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
     unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
  
     if (idx < cols && idy < rows) {
         unsigned int pos = idy * cols + idx;
         unsigned int trans_pos = idx * rows + idy;
  
         mat_out[trans_pos] = mat_in[pos];
     }
}


void transponerMatrix(char *h_mat_in, char *h_mat_out, unsigned int rows, unsigned int cols, size_t size){
    
    hipEvent_t start, stop;                                                                // --> Va en funcion transponerMatrix


    /* Pointer for host memory */
    
    /* Pointer for device memory */
 
    /* Allocate host and device memory */
    
 
    hipMalloc(&dev_mat_in, size);                                                      // --> Va en funcion transponerMatrix
    hipMalloc(&dev_mat_out, size);                                                     // --> Va en funcion transponerMatrix
 
    /* Fixed seed for illustration */
    srand(2047);                        // --> Va en funcion transponerMatrix


    // PRINTING ORIGINAL MATRIX
    // 
 
    hipEventCreate(&start); hipEventCreate(&stop);                                        // --> Va en funcion transponerMatrix
 
    /*------------------------ COMPUTATION ON GPU ----------------------------*/
 
    /* Host to device memory copy */
    hipMemcpy(dev_mat_in, h_mat_in, size, hipMemcpyHostToDevice);                     // --> Va en funcion transponerMatrix
 
    /* Set grid and block dimensions properly */
    unsigned int g_row = (rows + NUMBER_THREADS - 1) / NUMBER_THREADS;      // --> Va en funcion transponerMatrix
    unsigned int g_col = (cols + NUMBER_THREADS - 1) / NUMBER_THREADS;      // --> Va en funcion transponerMatrix
    dim3 bloques(g_col, g_row);                                                     // --> Va en funcion transponerMatrix
    dim3 hilos(NUMBER_THREADS, NUMBER_THREADS);                              // --> Va en funcion transponerMatrix
 
    hipEventRecord(start, 0);                                                              // --> Va en funcion transponerMatrix
 
    /* Launch kernel */
    transpose_gpu<<<bloques, hilos>>>(dev_mat_in, dev_mat_out, rows, cols);              // --> Va en funcion transponerMatrix
 
    hipEventRecord(stop, 0);                                                               // --> Va en funcion transponerMatrix
    hipEventSynchronize(stop);                                                             // --> Va en funcion transponerMatrix
    hipEventElapsedTime(&elapsed_time_ms, start, stop);                                    // --> Va en funcion transponerMatrix
 
    /* device to host copy */
    hipMemcpy(h_mat_out, dev_mat_out, size, hipMemcpyDeviceToHost);
 
}

int main(int argc, char **argv) {
 
    if (argc != 2) {
        fprintf(stderr, "Usage: %s matrix_size\n", argv[0]);
        fprintf(stderr, "       matrix_size is the number of rows and cols of the matrix\n");
        return EXIT_FAILURE;
    }
 
    unsigned int rows = atoi(argv[1]);
    unsigned int cols = rows;
 
    //Para poner tiempos en matriz
    FILE *fp;
    fp = fopen("./times.txt", "a+");

    char *h_mat_in, *h_mat_out;                                                             // --> Va en funcion transponerMatrix
    size_t size = rows * cols * sizeof(char);                                           // --> Va en funcion transponerMatrix
    h_mat_in = (char *)malloc(size);
    h_mat_out = (char *)malloc(size);
 
    /* Initialize host memory */
    for (unsigned int i = 0; i < rows; ++i) {
        for (unsigned int j = 0; j < cols; ++j) {
            h_mat_in[i * cols + j] = rand() % (rows * cols);
            //printf("%d\t", h_mat_in[i * cols + j]);
        }
        //printf("\n");
    }

    printMatrix(rows, cols, h_mat_in);

    transponerMatrix(h_mat_in, h_mat_out, rows, cols, size);

    printf("Time to transpose a matrix of %dx%d on GPU: %f ms.\n\n", rows, cols, elapsed_time_ms);
 
    //IMPRIMIR TIEMPOS EN ARCHIVO times.txt
    fprintf(fp, "[%d x %d] = %f ms.\n\n", rows, cols, elapsed_time_ms);
    fclose(fp);
 
    // PRINTING TRANSPOSED MATRIX
    printMatrix(rows, cols, h_mat_out);

    /* Free host and device memory */
    free(h_mat_in); free(h_mat_out); hipFree(dev_mat_in); hipFree(dev_mat_out);
}